#include <vector>

#include "caffe/layers/base_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(batch->label_);
    // Copy the labels.
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}

template <typename Dtype>
void ExBasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  ExBatch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  top[1]->ReshapeLike(batch->label_);
  top[2]->ReshapeLike(batch->dense_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
      top[1]->mutable_gpu_data());
  caffe_copy(batch->dense_.count(), batch->dense_.gpu_data(),
      top[2]->mutable_gpu_data());
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}

template <typename Dtype>
void RoiPrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  RoiBatch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  top[1]->ReshapeLike(batch->info_);
  if (output_roi_) top[2]->ReshapeLike(batch->roi_);
  if (output_labels_) top[3]->ReshapeLike(batch->label_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  caffe_copy(batch->info_.count(), batch->info_.gpu_data(),
      top[1]->mutable_gpu_data());
  if (output_roi_) {
    caffe_copy(batch->roi_.count(), batch->roi_.gpu_data(),
        top[2]->mutable_gpu_data());
  }
  if (output_labels_) {
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[3]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);
INSTANTIATE_LAYER_GPU_FORWARD(ExBasePrefetchingDataLayer);
INSTANTIATE_LAYER_GPU_FORWARD(RoiPrefetchingDataLayer);

}  // namespace caffe
